#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include "RSA_kernel.cu"
#define BUZZ_SIZE 10002

int p, q, n, t, flag, e[100], d[100], mm[BUZZ_SIZE], res[BUZZ_SIZE];
char msg[BUZZ_SIZE];
int prime(long int);
void generate_input(int);
void ce();
long int cd(long int);
void encrypt_cpu(void *ptr,int size);
void decrypt_cpu(void *ptr,int size);
void encrypt_gpu(void *ptr,int size);
void decrypt_gpu(void *ptr,int size);
int threadsPerBlock = 1024;
int blocksPerGrid;
time_t tt;
double time_encrypt_cpu, time_decrypt_cpu;
float time_encrypt_gpu = 0.0;
float time_decrypt_gpu = 0.0;
int prime(long int pr) {
	int j = sqrt(pr);
	for (int i = 2; i <= j; i++) {
		if (pr % i == 0)
			return 0;
	}
	return 1;
}

void ce() {
	int k;
	k = 0;
	for (int i = 2; i < t; i++) {
		if (t % i == 0)
			continue;
		flag = prime(i);
		if (flag == 1 && i != p && i != q) {
			e[k] = i;
			flag = cd(e[k]);
			if (flag > 0) {
				d[k] = flag;
				k++;
			}
			if (k == 99)
				break;
		}
	}
}

long int cd(long int x) {
	long int k = 1;
	while (1) {
		k = k + t;
		if (k % x == 0)
			return (k / x);
	}
}

void encrypt_cpu(void *h_data,int len) {
	double start_encrypt, end_encrypt;
	start_encrypt = clock();
	printf("CPU starts encrypting...\n");
	int pt, key = e[0], k;
	int *mm=(int *)h_data,*en=mm;
	printf("\ne=%d\n",key);
	// len /= sizeof(int);
	for(int i=0;i<len;i++){
		pt = mm[i];
		k = 1;
		for (int j = 0; j < key; j++) {
			k = k * pt;
			k = k % n;
		}
		en[i] = k;

	}
	end_encrypt = clock();
	time_encrypt_cpu = (double) (end_encrypt - start_encrypt) / CLOCKS_PER_SEC;
	printf("Encryption time taken by CPU: %f s\n", time_encrypt_cpu);
	/*
	 en[i] = -1;
	 printf("\nCPU ENCRYPTED MESSAGE IS\n");
	 for (i = 0; en[i] != -1; i++)
	 printf("%d ", en[i]);
	 */

	printf("Saving CPU encrypted file... ");
	// en[i] = -1;
	FILE *fp = fopen("encrypted_cpu.txt", "wb");
	if (fp != NULL) {
		for (int k = 0;k<len; k++) {
			fprintf(fp, "%d", en[k]);
		}
		fclose(fp);
		printf("done\n\n");
	}
}

void encrypt_gpu(void *d_data,int len) {
	hipEvent_t start_encrypt, stop_encrypt;
	int key = e[0];
	//printf("\nkey=%d, n=%d\n",key,n);
	hipSetDevice(1);
	int *dev_num=(int *)d_data, *dev_key, *dev_den;
	hipMalloc((void **) &dev_key, sizeof(int));
	hipMalloc((void **) &dev_den, sizeof(int));
	hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start_encrypt);
	hipEventCreate(&stop_encrypt);
	hipEventRecord(start_encrypt);
	printf("GPU starts encrypting...\n");
	rsa<<<blocksPerGrid, threadsPerBlock>>>(dev_num,dev_key,dev_den);
	hipEventRecord(stop_encrypt);
	hipEventSynchronize(stop_encrypt);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time_encrypt_gpu, start_encrypt, stop_encrypt);

	// hipMemcpy(res, dev_num, len * sizeof(int), hipMemcpyDeviceToHost);
	// hipFree(dev_num);
	hipFree(dev_key);
	hipFree(dev_den);
	time_encrypt_gpu /= 1000;
	printf("Encryption time taken by GPU: %f s\n", time_encrypt_gpu);

	/*
	 printf("\nGPU ENCRYPTED MESSAGE IS\n");
	 for (i = 0; i < len; i++)
	 printf("%d ", res[i]+96);
	 printf("\n");
	 */

	printf("Saving GPU encrypted file... ");
	// FILE *fp = fopen("encrypted_gpu.txt", "wb");
	// if (fp != NULL) {
	// 	for (i = 0; i < len; i++) {
	// 		fprintf(fp, "%d", res[i] + 96);
	// 	}
	// 	fclose(fp);
	// 	printf("done\n\n");
	// }
}

void decrypt_gpu(void *d_data,int len) {
	hipEvent_t start_decrypt, stop_decrypt;
	int key = d[0];
	//printf("\nkey=%d, n=%d\n",key,n);
	hipSetDevice(1);
	int *dev_num=(int*)d_data, *dev_key, *dev_den;
	hipMalloc((void **) &dev_key, sizeof(int));
	hipMalloc((void **) &dev_den, sizeof(int));
	hipMemcpy(dev_key, &key, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_den, &n, sizeof(int), hipMemcpyHostToDevice);

	hipEventCreate(&start_decrypt);
	hipEventCreate(&stop_decrypt);
	hipEventRecord(start_decrypt);
	printf("GPU starts decrypting...\n");
	rsa<<<blocksPerGrid, threadsPerBlock>>>(dev_num,dev_key,dev_den);
	hipEventRecord(stop_decrypt);
	hipEventSynchronize(stop_decrypt);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time_decrypt_gpu, start_decrypt, stop_decrypt);

	hipFree(dev_key);
	hipFree(dev_den);
	
	time_decrypt_gpu /= 1000;
	printf("Decryption time taken by GPU: %f s\n", time_decrypt_gpu);
	
	/*
	printf("\nGPU DECRYPTED MESSAGE IS\n");
	for (i = 0; i < len; i++)
	printf("%d ", res[i]+96);
	printf("\n");
	*/
	
}

void decrypt_cpu(void *h_data,int len) {
	double start_decrypt, end_decrypt;
	int *mm=(int *)h_data,*en=mm;
	start_decrypt = clock();
	printf("CPU starts decrypting...\n");
	long int key = d[0], k;
	printf("\nd=%d\n",key);
	// len/=sizeof(int);
	// #pragma omp parallel for
	for(int i=0;i<len;i++){
		int ct = en[i];
		k = 1;
		for (int j = 0; j < key; j++) {
			k = k * ct;
			k = k % n;
		}
		mm[i] = k;
		i++;
	}
	while (i<len) {
	}
	end_decrypt = clock();
	time_decrypt_cpu = (double) (end_decrypt - start_decrypt) / CLOCKS_PER_SEC;
	printf("Decryption time taken by CPU: %f s\n", time_decrypt_cpu);

	/*
	 m[i] = -1;
	 printf("\nCPU DECRYPTED MESSAGE IS\n");
	 for (i = 0; m[i] != -1; i++)
	 printf("%d ", m[i]);
	 printf("\n");
	 */

	printf("Saving CPU decrypted file... ");
	// FILE *fp = fopen("decrypted_cpu.txt", "wb");
	// if (fp != NULL) {
	// 	for (int k = 0; k<len; k++) {
	// 		fprintf(fp, "%c", mm[k]+96);
	// 	}
	// 	fprintf(fp, "\n");
	// 	fclose(fp);
	// 	printf("done\n\n");
	// }
}
