// #define _DEBUG
#include "RSA.cu"
#include <stdlib.h>
int numChars;
int main() {
    p= 74531;
    q= 37019;
    e=0x10001;
	d=985968293;
	// p= 126611;
    // q= 130643;
    // e=0x10001;
    // d=5621128193;

	// srand((unsigned) time(&tt));/* Intializes random number generator */
	// generate_input(100);

	// FILE *f = fopen("input.txt", "r");
	// if (f == NULL) {
	// 	perror("Error opening file");
	// 	return (1);
	// }
	// if (fgets(msg, BUZZ_SIZE, f) != NULL) {
	// 	//printf("String read: %s\n", msg);
	// 	printf("Reading input file...done(");
	// }
	// fclose(f);

	// numChars = strlen(msg) - 1;
	// msg[numChars] = '\0';
	n = p * q;
	srandom(2020);
	for(int i=0;i<10000;i++){
		mm[i]=random()%n;
	}
	numChars= 10000;
	printf("numChars: %d, n= %d)\n\n", numChars,n);
	blocksPerGrid = (numChars + threadsPerBlock - 1) / threadsPerBlock;
	FILE * F= fopen("input.txt","wb");
	if(F){
		for(int i=0;i<numChars;i++){
			fprintf(F,"%d ",mm[i]);
			if(i%10==9)fprintf(F,"\n");
		}
		fclose(F);
	}
	/*
	 printf("\nENTER MESSAGE\n");
	 fflush(stdin);
	 scanf("%s", msg);
	 numChars = strlen(msg);
	 blocksPerGrid =(numChars + threadsPerBlock - 1) / threadsPerBlock;
	 */

	// for (int i = 0; msg[i] != '\0'; i++)
	// 	mm[i] = msg[i] - 96;
	t = (p - 1) * (q - 1);
	// ce();
	/*
	 printf("\nPOSSIBLE VALUES OF e AND d ARE\n");
	 for (i = 0; i < j - 1; i++)
	 printf("\n%ld\t%ld", e[i], d[i]);
	 */

	encrypt_cpu(mm,numChars);
	decrypt_cpu(mm,numChars);
	FILE *fpc = fopen("decrypted_cpu.txt", "wb");
	if (fpc != NULL) {
		for (int i = 0; i < numChars; i++) {
			fprintf(fpc, "%u ", mm[i]);
			if(i%10==9)fprintf(fpc, "\n");
		}
		// fprintf(fpc, "\n");
		fclose(fpc);
		printf("done\n\n");
	}
	int * dev_num;
	hipMalloc((void **) &dev_num, numChars * sizeof(int));
	// hipMemcpy(dev_num, mm, numChars * sizeof(int), hipMemcpyHostToDevice);
	// encrypt_gpu(dev_num,numChars);
	// decrypt_gpu(dev_num,numChars);
	// printf("GPU encryption speed up: %f\n",
	// 		time_encrypt_cpu / time_encrypt_gpu);
	// printf("GPU decryption speed up: %f\n\n",
	// 		time_decrypt_cpu / time_decrypt_gpu);
	// hipMemcpy(res, dev_num, numChars * sizeof(int), hipMemcpyDeviceToHost);
	// printf("Saving GPU decrypted file... ");
	// FILE *fp = fopen("decrypted_gpu.txt", "wb");
	// if (fp != NULL) {
	// 	for (int i = 0; i < numChars; i++) {
	// 		fprintf(fp, "%u ", res[i]);
	// 		if(i%10==9)fprintf(fp, "\n");
	// 	}
	// 	fclose(fp);
	// 	printf("done\n\n");
	// }
	hipFree(dev_num);
	return 0;
}

void generate_input(int size) {
	printf("\nGenerating input file... ");
	FILE *fp = fopen("input.txt", "wb");
	if (fp != NULL) {
		for (int k = 0; k < size; k++) {
			int r = rand() % 26;
			fprintf(fp, "%c", r + 97);
		}
		fprintf(fp, "\n");
		fclose(fp);
		printf("done\n");
	}
}

