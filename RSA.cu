#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <omp.h>
#include "RSA_kernel.cu"
#define BUZZ_SIZE 10002

unsigned long int p, q, n, t, flag, e, d;
unsigned int mm[BUZZ_SIZE], res[BUZZ_SIZE];
void generate_input(int);
void encrypt_cpu(void *ptr,int size);
void decrypt_cpu(void *ptr,int size);
void encrypt_gpu(void *ptr,int size);
void decrypt_gpu(void *ptr,int size);
int threadsPerBlock = 1024;
int blocksPerGrid=64;
time_t tt;
double time_encrypt_cpu, time_decrypt_cpu;
float time_encrypt_gpu = 0.0;
float time_decrypt_gpu = 0.0;
int prime(long int pr) {
	int j = sqrt(pr);
	for (int i = 2; i <= j; i++) {
		if (pr % i == 0)
			return 0;
	}
	return 1;
}

void encrypt_cpu(void *h_data,int len) {
	double start_encrypt, end_encrypt;
	start_encrypt = clock();
	printf("CPU starts encrypting...\n");
	// unsigned long int  key = e, k;
	unsigned int *mm=(unsigned int *)h_data,*en=mm;
	// printf("\ne=%d\n",key);
	#ifdef _DEBUG
	printf("n%u\n\n\n\n",n);
	#endif

	// len /= sizeof(int);
	#pragma omp parallel for
	for(int i=0;i<len;i++){
		unsigned long key=e,k=1,exp=mm[i]%n;
		while(key){
			if(key%2){
				k*=exp;
				k%=n;
			}
			key/=2;
			exp*=exp;
			exp%=n;
		}
		en[i] = (unsigned int)k;		
		#ifdef _DEBUG
		if(k<0)printf("en_ERROR!!!!!!!!!!!!\n\n\n\n");
		#endif
	}
	end_encrypt = clock();
	time_encrypt_cpu = (double) (end_encrypt - start_encrypt) / CLOCKS_PER_SEC;
	printf("Encryption time taken by CPU: %f s\n", time_encrypt_cpu);
	/*
	 en[i] = -1;
	 printf("\nCPU ENCRYPTED MESSAGE IS\n");
	 for (i = 0; en[i] != -1; i++)
	 printf("%d ", en[i]);
	 */

	printf("Saving CPU encrypted file... ");
	// FILE *fp = fopen("encrypted_cpu.txt", "wb");
	// if (fp != NULL) {
	// 	for (int k = 0;k<len; k++) {
	// 		fprintf(fp, "%d", en[k]);
	// 	}
	// 	fclose(fp);
	// 	printf("done\n\n");
	// }
}

void encrypt_gpu(void *d_data,int len) {
	hipEvent_t start_encrypt, stop_encrypt;
	unsigned long int key = e;
	//printf("\nkey=%d, n=%d\n",key,n);
	hipSetDevice(1);
	unsigned int *dev_num=(unsigned int *)d_data;
	unsigned long *dev_key, *dev_den;
	hipMalloc((void **) &dev_key, sizeof(long int));
	hipMalloc((void **) &dev_den, sizeof(long int));
	hipMemcpy(dev_key, &key, sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(dev_den, &n, sizeof(long int), hipMemcpyHostToDevice);

	hipEventCreate(&start_encrypt);
	hipEventCreate(&stop_encrypt);
	hipEventRecord(start_encrypt);
	printf("GPU starts encrypting...\n");
	blocksPerGrid=(len+threadsPerBlock-1)/threadsPerBlock;
	rsa<<<blocksPerGrid, threadsPerBlock>>>(dev_num,dev_key,dev_den,len);
	hipEventRecord(stop_encrypt);
	hipEventSynchronize(stop_encrypt);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time_encrypt_gpu, start_encrypt, stop_encrypt);

	// hipMemcpy(res, dev_num, len * sizeof(int), hipMemcpyDeviceToHost);
	// hipFree(dev_num);
	hipFree(dev_key);
	hipFree(dev_den);
	time_encrypt_gpu /= 1000;
	printf("Encryption time taken by GPU: %f s\n", time_encrypt_gpu);

	/*
	 printf("\nGPU ENCRYPTED MESSAGE IS\n");
	 for (i = 0; i < len; i++)
	 printf("%d ", res[i]+96);
	 printf("\n");
	 */

	printf("Saving GPU encrypted file... ");
	// FILE *fp = fopen("encrypted_gpu.txt", "wb");
	// if (fp != NULL) {
	// 	for (int i = 0; i < len; i++) {
	// 		fprintf(fp, "%d", res[i] + 96);
	// 	}
	// 	fclose(fp);
	// 	printf("done\n\n");
	// }
}

void decrypt_gpu(void *d_data,int len) {
	hipEvent_t start_decrypt, stop_decrypt;
	unsigned long int key = d;
	//printf("\nkey=%d, n=%d\n",key,n);
	hipSetDevice(1);
	unsigned int *dev_num=(unsigned int*)d_data;
	unsigned long *dev_key, *dev_den;
	hipMalloc((void **) &dev_key, sizeof(long int));
	hipMalloc((void **) &dev_den, sizeof(long int));
	hipMemcpy(dev_key, &key, sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(dev_den, &n, sizeof(long int), hipMemcpyHostToDevice);

	hipEventCreate(&start_decrypt);
	hipEventCreate(&stop_decrypt);
	hipEventRecord(start_decrypt);
	printf("GPU starts decrypting...\n");
	blocksPerGrid=(len+threadsPerBlock-1)/threadsPerBlock;
	rsa<<<blocksPerGrid, threadsPerBlock>>>(dev_num,dev_key,dev_den,len);
	hipEventRecord(stop_decrypt);
	hipEventSynchronize(stop_decrypt);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time_decrypt_gpu, start_decrypt, stop_decrypt);

	hipFree(dev_key);
	hipFree(dev_den);
	
	time_decrypt_gpu /= 1000;
	printf("Decryption time taken by GPU: %f s\n", time_decrypt_gpu);
}

void decrypt_cpu(void *h_data,int len) {
	double start_decrypt, end_decrypt;
	unsigned int *mm=(unsigned int *)h_data,*en=mm;
	start_decrypt = clock();
	printf("CPU starts decrypting...\n");
	// unsigned long int  ct, key = d, k;
	// printf("\nd=%d\n",key);
	// len/=sizeof(int);
	#pragma omp parallel for
	for(int i=0;i<len;i++) {
		unsigned long ct = en[i]%n,k = 1,key=d;
		while(key){
			if(key%2==1){
				k*=ct;
				k%=n;
			}
			key/=2;
			ct*=ct;
			ct%=n;
		}
		mm[i] = (unsigned int)k;
		#ifdef _DEBUG
		if(k<0)printf("decrypt_ERROR!!!!!!!!!!!!\n\n\n\n");
		#endif
	}
	end_decrypt = clock();
	time_decrypt_cpu = (double) (end_decrypt - start_decrypt) / CLOCKS_PER_SEC;
	printf("Decryption time taken by CPU: %f s\n", time_decrypt_cpu);
	printf("Saving CPU decrypted file... ");
}
